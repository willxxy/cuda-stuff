#include <stdio.h>
#include <hip/hip_runtime.h>
int main(){
    int n;
    hipGetDeviceCount(&n);
    for(int i=0; i<n; ++i){
        hipDeviceProp_t p;
        hipGetDeviceProperties(&p, i);
        printf("GPU %d: %s – SMs: %d\n", i, p.name, p.multiProcessorCount);
    }
    return 0;
}

